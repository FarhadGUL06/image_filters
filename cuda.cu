#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cstdint>


// Functions in this file

const uint8_t rgb_max_value = 255;
const int filterHeight = 11;
const int filterWidth = 11;

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int height, width;
    PPMPixel *data;
} PPMImage;


// Headers for functions
// Utils
PPMImage *apply_filter(PPMImage *image, double **filter, int filterHeight, int filterWidth, double factor, double bias);


// PPM IO
PPMImage *read_PPM(const char *filename);
void write_PPM(const char *filename, PPMImage *img);

// Filters
PPMImage *blur_PPM(PPMImage *image);
PPMImage *gaussian_blur_PPM(PPMImage *image);
PPMImage *motion_blur_PPM(PPMImage *image);
PPMImage *sharpen_PPM(PPMImage *image);
PPMImage *extreme_sharpen_PPM(PPMImage *image);
PPMImage *find_edges_PPM(PPMImage *image);
PPMImage *emboss_PPM(PPMImage *image);
PPMImage *extreme_emboss_PPM(PPMImage *image);
PPMImage *random_PPM(PPMImage *image);

// Array containing filters and id for each filter
static PPMImage *(*filters[])(PPMImage *) = {
    blur_PPM,
    gaussian_blur_PPM,
    motion_blur_PPM,
    sharpen_PPM,
    extreme_sharpen_PPM,
    find_edges_PPM,
    emboss_PPM,
    extreme_emboss_PPM,
    random_PPM
};

// Struct for payload
struct thread_payload {
    size_t thread_id;
    PPMImage **images;
    PPMImage *(**to_apply)(PPMImage *);
    int number_of_filters;
    int start;
    int end;
};

__global__ void thread_pixel(PPMPixel *d_data,
                             double *d_filter, int *d_fi, int *d_fj,
                            PPMPixel *d_result_data, double *d_factor,
                             double *d_bias, int *d_width, int *d_height) {

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index > *d_width * *d_height) {
        return;
    }

    int d_i = index % *d_width;
    int d_j = index / *d_width;

    double red = 0;
    double green = 0;
    double blue = 0;
    for (int k = 0; k < *d_fi; k++) {
        for (int l = 0; l < *d_fj; l++) {
            int index_i = d_i - *d_fi / 2 + k;
            int index_j = d_j - *d_fj / 2 + l;

            if (index_i >= 0 && index_i < *d_height && index_j >= 0 &&
                index_j <*d_width) {
                red += d_data[index_j *  *d_height + index_i].red *
                       d_filter[k * *d_fi + l];
                green +=
                    d_data[index_j * *d_height + index_i].green *
                    d_filter[k * *d_fi + l];
                blue +=
                    d_data[index_j * *d_height + index_i].blue *
                    d_filter[k * *d_fi + l];
                printf("%lf %lf %lf\n", red, green, blue);
            }
        }
    }
    d_result_data[d_j * *d_height + d_i].red =
        min(max((int)(*d_factor * red + *d_bias), 0), 255);
    d_result_data[d_j * *d_height + d_i].green =
        min(max((int)(*d_factor * green + *d_bias), 0), 255);
    d_result_data[d_j * *d_height + d_i].blue =
        min(max((int)(*d_factor * blue + *d_bias), 0), 255);
        
    //printf("All parameters: %d %d %d %d %d %d %lf %lf %d\n", d_i, d_j, *d_fi, *d_fj, *d_width, *d_height, *d_factor, *d_bias, index);
}

void read_all_images(PPMImage **images, int number_of_images) {
    for (int i = 0; i < number_of_images; i++) {
        char filename[50];
        sprintf(filename, "images/ppm_before/image%d.ppm", i);
        images[i] = read_PPM(filename);
    }
}

void write_all_images(PPMImage **images, int number_of_images) {
    for (int i = 0; i < number_of_images; i++) {
        char filename[50];
        sprintf(filename, "images/ppm_after/image%d.ppm", i);
        write_PPM(filename, images[i]);
    }
}

void free_all_images(PPMImage **images, int number_of_images,
                     PPMImage *(**to_apply)(PPMImage *)) {
    for (int i = 0; i < number_of_images; i++) {
        free(images[i]->data);
        free(images[i]);
    }
    free(images);
    free(to_apply);
}

PPMImage *(**read_all_filters(int *number_of_filters))(PPMImage *) {
    scanf("%d", number_of_filters);
    PPMImage *(**to_apply)(PPMImage *) = (PPMImage * (**)(PPMImage *))
        malloc(sizeof(PPMImage * (*)()) * (*number_of_filters + 1));

    for (int i = 0; i < *number_of_filters; i++) {
        int filter_id;
        scanf("%d", &filter_id);

        to_apply[i] = filters[filter_id];
    }

    return to_apply;
}


void apply_all_filters(PPMImage **images, int number_of_images,
                       PPMImage *(**to_apply)(PPMImage *),
                       int number_of_filters) {

    for (int i = 0; i < number_of_images; i++) {
        for (int j = 0; j < number_of_filters; j++) {
            images[i] = to_apply[j](images[i]);
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: ./cuda <number_of_images> <num_threads>\n");
        return 1;
    }
    const int number_of_images = atoi(argv[1]);
    if (number_of_images <= 0 || number_of_images > 50) {
        printf("Invalid number of images\n");
        return 1;
    }

    int number_of_filters;
    PPMImage **images =
        (PPMImage **)malloc(sizeof(PPMImage *) * number_of_images);
    read_all_images(images, number_of_images);

    PPMImage *(**to_apply)(PPMImage *) = read_all_filters(&number_of_filters);

    apply_all_filters(images, number_of_images, to_apply, number_of_filters);

    write_all_images(images, number_of_images);
    free_all_images(images, number_of_images, to_apply);
    return 0;
}


PPMImage *apply_filter(PPMImage *image, double **filter, int filterHeight, int filterWidth, double factor, double bias) {
    PPMPixel *d_data;
    PPMPixel *d_result_data;
    double *d_filter;
    double *d_factor;
    double *d_bias;
    int *d_width;
    int *d_height;
    int *d_i;
    int *d_j;
    int *d_fi;
    int *d_fj;

    // cudaMalloc
    hipMalloc(&d_data, sizeof(PPMPixel) * image->height * image->width);
    hipMalloc(&d_result_data, sizeof(PPMPixel) * image->height * image->width);
    hipMalloc(&d_factor, sizeof(double));
    hipMalloc(&d_bias, sizeof(double));
    hipMemset(d_factor, 0, sizeof(double));
    hipMemset(d_bias, 0, sizeof(double));
    hipMalloc(&d_width, sizeof(int));
    hipMalloc(&d_height, sizeof(int));
    hipMalloc(&d_i, sizeof(int));
    hipMalloc(&d_j, sizeof(int));
    hipMalloc(&d_fi, sizeof(int));
    hipMalloc(&d_fj, sizeof(int));

    // cudaMemcpy
    printf("Bias and factor %f %f\n", bias, factor);
    hipMemcpy(d_data, image->data, sizeof(PPMPixel) * image->height * image->width, hipMemcpyHostToDevice);
    hipMemcpy(d_factor, &factor, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, &bias, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &image->width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_height, &image->height, sizeof(int), hipMemcpyHostToDevice);

    // d_filter combine them into one array
    double *filter_array = (double *)malloc(sizeof(double) * filterHeight * filterWidth);
    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter_array[i * filterHeight + j] = filter[i][j];
        }
    }
    hipMalloc(&d_filter, sizeof(double) * filterHeight * filterWidth);
    hipMemcpy(d_filter, filter_array, sizeof(double) * filterHeight * filterWidth, hipMemcpyHostToDevice);

    // cudaMemsetNull
    hipMemset(d_result_data, 0, sizeof(PPMPixel) * image->height * image->width);

    // Kernel
    int blockSize = 256;
    int numBlocks = 65535;
    
    for (int i = 0; i < image->height; i++) {
        for (int j = 0; j < image->width; j++) {
            hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_j, &j, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_fi, &filterHeight, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_fj, &filterWidth, sizeof(int), hipMemcpyHostToDevice);
            thread_pixel<<<numBlocks, blockSize>>>(d_data, d_filter, d_fi, d_fj, d_result_data, d_factor, d_bias, d_width, d_height);
        }
    }

    // cudaMemcpy
    hipMemcpy(image->data, d_result_data, sizeof(PPMPixel) * image->height * image->width, hipMemcpyDeviceToHost);


}

PPMImage *read_PPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;

    // open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    // read image format
    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    // check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    // alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    // check for comments
    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    // read image size information
    if (fscanf(fp, "%d %d", &img->height, &img->width) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    // read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    // check rgb component depth
    if (rgb_comp_color != rgb_max_value) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    // memory allocation for pixel data
    img->data = (PPMPixel *)malloc(img->height * img->width * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    // read pixel data from file
    if (fread(img->data, 3 * img->height, img->width, fp) != img->width) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void write_PPM(const char *filename, PPMImage *img) {
    FILE *fp;
    // open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    // write the header file
    // image format
    fprintf(fp, "P6\n");

    // image size
    fprintf(fp, "%d %d\n", img->height, img->width);

    // rgb component depth
    fprintf(fp, "%d\n", rgb_max_value);

    // pixel data
    fwrite(img->data, 3 * img->height, img->width, fp);
    fclose(fp);
}

double compute_factor(double copy[filterHeight][filterWidth]) {
    int sum = 0;
    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            sum += copy[i][j];
        }
    }
    return 1.0 / sum;
}

PPMImage *blur_PPM(PPMImage *image) {
    int bias = 0;
    double factor;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
        {0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0}, {0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 0, 1, 1, 1, 1, 1, 0, 0, 0}, {0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0}, {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0}, {0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {0, 0, 0, 1, 1, 1, 1, 1, 0, 0, 0}, {0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0},
    };

    factor = compute_factor(copy);

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *gaussian_blur_PPM(PPMImage *image) {
    int bias = 0;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
        {1, 4, 8, 10, 10, 8, 4, 1, 0, 0, 0},
        {4, 16, 32, 40, 40, 32, 16, 4, 0, 0, 0},
        {8, 32, 64, 80, 80, 64, 32, 8, 0, 0, 0},
        {10, 40, 80, 100, 100, 80, 40, 10, 0, 0, 0},
        {10, 40, 80, 100, 100, 80, 40, 10, 0, 0, 0},
        {8, 32, 64, 80, 80, 64, 32, 8, 0, 0, 0},
        {4, 16, 32, 40, 40, 32, 16, 4, 0, 0, 0},
        {1, 4, 8, 10, 10, 8, 4, 1, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    };

    factor = compute_factor(copy);

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *motion_blur_PPM(PPMImage *image) {
    int bias = 0;
    double factor;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
        {1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}, {0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0}, {0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0}, {0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0}, {0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0}, {0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1},
    };

    factor = compute_factor(copy);

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *sharpen_PPM(PPMImage *image) {
    int bias = 0;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, -1, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, 9, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, -1, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    };

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *extreme_sharpen_PPM(PPMImage *image) {
    int bias = 0;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, -7, 1, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        };

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *find_edges_PPM(PPMImage *image) {
    int bias = 0;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
         {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, -1, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, 8, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, -1, -1, -1, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    };

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *emboss_PPM(PPMImage *image) {
    const int filterHeight = 3;
    const int filterWidth = 3;

    int bias = 128;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {-1, -1, 0, -1, 0, 1, 0, 1, 1};

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *extreme_emboss_PPM(PPMImage *image) {
    const int filterHeight = 5;
    const int filterWidth = 5;

    int bias = 128;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {-1, -1, -1, -1, 0, -1, -1, -1, 0,
                                              1,  -1, -1, 0,  1, 1,  -1, 0,  1,
                                              1,  1,  0,  1,  1, 1,  1};

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}

PPMImage *random_PPM(PPMImage *image) {
    int bias = 0;
    double factor = 1.0;
    double **filter = (double **)malloc(filterHeight * sizeof(double *));
    for (int i = 0; i < filterHeight; i++) {
        filter[i] = (double *)malloc(filterWidth * sizeof(double));
    }
    double copy[filterHeight][filterWidth] = {
         {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, 121, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
        {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    };

    for (int i = 0; i < filterHeight; i++) {
        for (int j = 0; j < filterWidth; j++) {
            filter[i][j] = copy[i][j];
        }
    }
    PPMImage *result =
        apply_filter(image, filter, filterHeight, filterWidth, factor, bias);
    return result;
}



